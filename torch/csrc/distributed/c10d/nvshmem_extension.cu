#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/nvshmem_extension.cuh>

#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/SymmetricMemory.hpp>

#include <cuda_awbarrier_primitives.h>
#include <nvshmem.h>

namespace c10d::nvshmem_extension {

const std::string store_comm_prefix = "nvshmem_extension";
static size_t store_comm_seq_id = 0;

template <typename T>
std::vector<T> store_all_gather(
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size,
    T val) {
  static_assert(std::is_trivially_copyable_v<T>);

  std::vector<std::string> peer_keys;
  for (int r = 0; r < world_size; ++r) {
    std::ostringstream oss;
    oss << store_comm_prefix << "/" << store_comm_seq_id << "/" << r;
    peer_keys.push_back(oss.str());
  }
  ++store_comm_seq_id;

  {
    std::vector<uint8_t> payload(
        reinterpret_cast<uint8_t*>(&val),
        reinterpret_cast<uint8_t*>(&val) + sizeof(T));
    store->set(peer_keys[rank], payload);
  }

  std::vector<T> peer_vals;
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      peer_vals.push_back(val);
      continue;
    }
    store->wait({peer_keys[r]});
    auto payload = store->get(peer_keys[r]);
    TORCH_CHECK(payload.size() == sizeof(T));
    T peer_val{};
    std::memcpy(&peer_val, payload.data(), sizeof(T));
    peer_vals.push_back(peer_val);
  }
  return peer_vals;
}

void maybe_initialize_env_vars() {
  auto nccl_socket_if_name = c10::utils::get_env("NCCL_SOCKET_IFNAME");
  auto nccl_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nccl_ib_gid_index = c10::utils::get_env("NCCL_IB_GID_INDEX");
  auto nvshmem_socket_if_name =
      c10::utils::get_env("NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME");
  auto nvshmem_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nvshmem_ib_gid_index = c10::utils::get_env("NVSHMEM_IB_GID_INDEX");

  if (!nvshmem_socket_if_name.has_value() && nccl_socket_if_name.has_value()) {
    c10::utils::set_env(
        "NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME", nccl_socket_if_name->c_str());
  }
  if (!nvshmem_hca_list.has_value() && nccl_hca_list.has_value()) {
    c10::utils::set_env("NVSHMEM_ENABLE_NIC_PE_MAPPING", "1");
    c10::utils::set_env("NVSHMEM_HCA_LIST", nccl_hca_list->c_str());
  }
  if (!nvshmem_ib_gid_index.has_value() && nccl_ib_gid_index.has_value()) {
    c10::utils::set_env("NVSHMEM_IB_GID_INDEX", nccl_ib_gid_index->c_str());
  }
}

void initialize_nvshmem_with_store(
    c10::intrusive_ptr<c10d::Store> store,
    int rank,
    int world_size) {
  static bool is_initialized = false;
  if (is_initialized) {
    return;
  }

  maybe_initialize_env_vars();

  nvshmemx_uniqueid_t unique_id;
  TORCH_CHECK(
      nvshmemx_get_uniqueid(&unique_id) == 0, "nvshmemx_get_uniqueid failed");

  // Using an existing store_all_gather due to laziness.
  // TODO(yifu): should use broadcast
  auto unique_ids = store_all_gather(store, rank, world_size, unique_id);

  nvshmemx_init_attr_t attr;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, &unique_ids[0], &attr);

  TORCH_CHECK(
      nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr) == 0,
      "nvshmemx_init_attr failed");

  is_initialized = true;
}

void* nvshmem_malloc(size_t size) {
  return ::nvshmem_malloc(size);
}

void* nvshmem_ptr(const void* dest, int pe) {
  return ::nvshmem_ptr(dest, pe);
}

std::unordered_map<std::string, nvshmem_team_t> group_name_to_team_;

nvshmem_team_t group_to_team(
    const std::string& group_name,
    const std::vector<int>& global_ranks) {
  auto it = group_name_to_team_.find(group_name);
  if (it != group_name_to_team_.end()) {
    return it->second;
  }
  TORCH_CHECK(global_ranks.size() > 1);
  int stride = global_ranks[1] - global_ranks[0];
  for (size_t r = 1; r < global_ranks.size(); ++r) {
    TORCH_CHECK(global_ranks[r] - global_ranks[r - 1] == stride);
  }

  nvshmem_team_t team;
  TORCH_CHECK(
      nvshmem_team_split_strided(
          NVSHMEM_TEAM_WORLD,
          global_ranks[0],
          stride,
          global_ranks.size(),
          nullptr,
          0,
          &team) == 0);
  group_name_to_team_[group_name] = team;
  TORCH_CHECK(team != NVSHMEM_TEAM_INVALID);
  return team;
}

at::Tensor nvshmem_broadcast(at::Tensor& input, const std::string& group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());
  void* buffer_ptr = input_hdl->get_buffer_ptrs()[rank];

  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_broadcastmem_on_stream(team, buffer_ptr, buffer_ptr, input_hdl->get_buffer_size(), 0, stream);
  return input;
}

// TODO: lock-free ring buf
template <typename T, int N>
struct ring_buf {
  T data[N];
  uint32_t head;
  uint32_t tail;
  uint32_t lock;
};

template <typename T, int N>
__device__ void ring_buf_init(ring_buf<T, N>* buf) {
  buf->head = 0;
  buf->tail = 0;
  buf->lock = 0;
}

template <typename T, int N>
__device__ void ring_buf_lock(ring_buf<T, N>* buf) {
  while (atomicCAS(&buf->lock, 0, 1) != 0) {
    __nanosleep(100);
  }
}

template <typename T, int N>
__device__ void ring_buf_unlock(ring_buf<T, N>* buf) {
  while (atomicCAS(&buf->lock, 1, 0) != 1) {
    __nanosleep(100);
  }
}

template <typename T, int N>
__device__ bool ring_buf_try_push(ring_buf<T, N>* buf, const T& value) {
  ring_buf_lock(buf);

  int new_head = (buf->head + 1) % N;
  if (new_head == buf->tail) {
    // Buffer is full
    ring_buf_unlock(buf);
    return false;
  }

  // Store the data and advance the head
  buf->data[buf->head] = value;
  buf->head = new_head;
  ring_buf_unlock(buf);
  return true;
}

template <typename T, int N>
__device__ bool ring_buf_try_pop(ring_buf<T, N>* buf, T& out) {
  ring_buf_lock(buf);

  if (buf->tail == buf->head) {
    // Buffer is empty
    ring_buf_unlock(buf);
    return false;
  }

  // Retrieve the data and advance the tail
  out = buf->data[buf->tail];
  buf->tail = (buf->tail + 1) % N;
  ring_buf_unlock(buf);
  return true;
}

template <typename T, int N>
__device__ void ring_buf_push(ring_buf<T, N>* buf, const T& value) {
  while (!ring_buf_try_push(buf, value)) {
    __nanosleep(100);
  }
}

template <typename T, int N>
__device__ T ring_buf_pop(ring_buf<T, N>* buf) {
  T value;
  while (!ring_buf_try_pop(buf, value)) {
    __nanosleep(100);
  }
  return value;
}

//         |  |  |
//         v  v  v
//        +--+--+--+--+
// rank 0 |  |  |  |  |
//        +--+--+--+--+
//            |  |  |
//            v  v  v
//        +--+--+--+--+
// rank 1 |  |  |  |  |
//        +--+--+--+--+
//         |     |  |
//         v     v  v
//        +--+--+--+--+
// rank 2 |  |  |  |  |
//        +--+--+--+--+
//         |  |     |
//         v  v     v
//        +--+--+--+--+
// rank 3 |  |  |  |  |
//        +--+--+--+--+
//         |  |  |
//         v  v  v
template <typename T, bool debug>
__global__ void nvshmem_all_reduce_kernel(
    T* input_ptr,
    T* output_ptr,
    size_t numel,
    uint64_t* signal_pad_ptr,
    int rank,
    int world_size,
    int* rank_to_global_rank,
    nvshmem_team_t team) {
  __shared__ ring_buf<int, 128> acc_queue;
  __shared__ int acc_split_idx;

  if (threadIdx.x == 0) {
    ring_buf_init(&acc_queue);
  }
  __syncthreads();

  constexpr int warp_size = 32;
  const int warp_idx = threadIdx.x / warp_size;
  const size_t split_size = numel / world_size;  // TODO: handle unaligned
  const size_t chunk_size = split_size / gridDim.x; // TODO: handle unaligned

  // Split the signal pad among blocks
  uint64_t* split_signals = &signal_pad_ptr[blockIdx.x * world_size];

  if (warp_idx == 0) {
    // ==================
    // Communication warp
    // ==================
    const int thread_idx = threadIdx.x;
    const int next_global_rank = rank_to_global_rank[(rank + 1) % world_size];
    if (thread_idx != 0) {
      return;
    }

    {
      const int split_idx = (rank + world_size - 1) % world_size;
      const size_t split_begin = split_idx * split_size;
      const size_t chunk_begin = split_begin + blockIdx.x * chunk_size;
      nvshmem_int_put_signal_nbi(
          output_ptr + chunk_begin,
          input_ptr + chunk_begin,
          chunk_size,
          &split_signals[split_idx],
          1,
          NVSHMEM_SIGNAL_SET,
          next_global_rank);
    }

    int received = 0, forwarded = 1;
    while (true) {
      for (int split_idx = 0;
           split_idx < world_size && received != world_size - 1;
           ++split_idx) {
        if (nvshmem_uint64_test(&split_signals[split_idx], NVSHMEM_CMP_EQ, 1)) {
          ring_buf_push(&acc_queue, split_idx);
          split_signals[split_idx] = 0;
          received += 1;
        }
      }

      for (int split_idx = 0;
           split_idx < world_size && forwarded != world_size - 1;
           ++split_idx) {
        if (nvshmem_uint64_test(&split_signals[split_idx], NVSHMEM_CMP_EQ, 2)) {
          const int split_begin = split_idx * split_size;
          const int chunk_begin = split_begin + blockIdx.x * chunk_size;
          nvshmem_int_put_signal_nbi(
              output_ptr + chunk_begin,
              output_ptr + chunk_begin,
              chunk_size,
              &split_signals[split_idx],
              1,
              NVSHMEM_SIGNAL_SET,
              next_global_rank);
          split_signals[split_idx] = 0;
          forwarded += 1;
        }
      }

      if (forwarded == world_size - 1 && received == world_size - 1) {
        nvshmem_quiet();
        break;
      }
    }
  } else {
    // ===============
    // Reduction wraps
    // ===============
    const int thread_idx = threadIdx.x - warp_size;
    const int num_threads = blockDim.x - warp_size;

    for (int step = 0; step < world_size - 1; ++step) {
      if (thread_idx == 0) {
        int split_idx = ring_buf_pop(&acc_queue);
        acc_split_idx = split_idx;
      }
      asm volatile("bar.sync 0, 512;" : : : "memory");

      const int split_idx = acc_split_idx;
      const size_t split_begin = split_idx * split_size;
      const size_t chunk_begin = split_begin + blockIdx.x * chunk_size;

      for (size_t offset = chunk_begin;
           offset < std::min(chunk_begin + chunk_size, numel);
           offset += num_threads) {
        if (offset + thread_idx < numel) {
          output_ptr[offset + thread_idx] =
              output_ptr[offset + thread_idx] + input_ptr[offset + thread_idx];
        }
      }
      asm volatile("bar.sync 0, 512;" : : : "memory");

      if (thread_idx == 0) {
        if (split_idx != rank) {
          split_signals[split_idx] = 2;
        }
      }
    }
  }
}

at::Tensor nvshmem_reduce_scatter_out(
    at::Tensor& input,
    std::string group_name,
    at::Tensor& out) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  size_t numel = input.numel();
  void* signal_pad_ptr = input_hdl->get_signal_pad_ptrs()[rank];
  void* rank_to_global_rank = input_hdl->get_rank_to_global_rank_dev();
  void* args[] = {
      &input_ptr,
      &output_ptr,
      &numel,
      &signal_pad_ptr,
      &rank,
      &world_size,
      &rank_to_global_rank,
      &team};

  dim3 grid_dim(32), block_dim(544);
  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_barrier_on_stream(team, stream);
  nvshmemx_collective_launch(
      (const void*)nvshmem_all_reduce_kernel<int, true>,
      grid_dim,
      block_dim,
      args,
      0,
      stream);
  nvshmemx_barrier_on_stream(team, stream);
  return out;
}

#define THREADS_PER_BLOCK 512

__global__ void sendrecv(float *send_data, float *recv_data, int num_elems, int mype,
                                     int npes) {
    int peer = (mype + 1) % npes;
    int block_offset = blockIdx.x * blockDim.x;
    // All threads in a block call the API with the same arguments
    nvshmemx_float_put_block(recv_data + block_offset, send_data + block_offset,
                             min(blockDim.x, num_elems - block_offset),
                             peer);
}

at::Tensor nvshmem_sendrecv(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();

  float* input_ptr = (float*)(input_hdl->get_buffer_ptrs()[rank]);
  float* output_ptr = (float*)(out_hdl->get_buffer_ptrs()[rank]);
  size_t numel = input.numel();

  assert(numel % THREADS_PER_BLOCK == 0); /* for simplicity */
  int num_blocks = numel / THREADS_PER_BLOCK;

  sendrecv<<<num_blocks, THREADS_PER_BLOCK>>>(input_ptr, output_ptr, numel, rank, world_size);
  return out;
}

at::Tensor nvshmem_all_to_all(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = (float*)(input_hdl->get_buffer_ptrs()[rank]);
  void* output_ptr = (float*)(out_hdl->get_buffer_ptrs()[rank]);
  assert input_hdl->get_buffer_size() % world_size == 0;
  size_t bytes_per_rank = input_hdl->get_buffer_size() / world_size;

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());
  nvshmemx_alltoallmem_on_stream(team, output_ptr, input_ptr, bytes_per_rank, stream);
  return out;
}

} // namespace c10d::nvshmem_extension
